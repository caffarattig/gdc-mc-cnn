#include "hip/hip_runtime.h"
extern "C" {
   #include "lua.h"
   #include "lualib.h"
   #include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <stdint.h>
#include <unistd.h>
#include <png++/image.hpp>

#define TB 128

THCState* getCutorchState(lua_State* L)
{
   lua_getglobal(L, "cutorch");
   lua_getfield(L, -1, "getState");
   lua_call(L, 0, 1);
   THCState *state = (THCState*) lua_touserdata(L, -1);
   lua_pop(L, 2);
   return state;
}

void checkCudaError(lua_State *L) {
   hipError_t status = hipPeekAtLastError();
   if (status != hipSuccess) {
      luaL_error(L, hipGetErrorString(status));
   }
}

__global__ void outlier_detection(float *d0, float *d1, float *outlier, int size, int dim3, float *conf1, float *conf2, int disp_max, float t1, float t2)
{
   int id = blockIdx.x * blockDim.x + threadIdx.x;
   if (id < size) {
      int x = id % dim3;
      int d0i = d0[id];
      if (x - d0i < 0) {
         //assert(0);
         outlier[id] = 1;
      } else if ((abs(d0[id] - d1[id - d0i]) < 1.1) 
            || (conf1[id] > t1
               && (conf1[id] - conf2[id- d0i] > t2)
            )){
         outlier[id] = 0; /* match */
      } else {
         outlier[id] = 1; /* occlusion */
         for (int d = 0; d < disp_max; d++) {
            if (x - d >= 0 && abs(d - d1[id - d]) < 1.1) {
               outlier[id] = 2; /* mismatch */
               break;
            }
         }
      }
   }
}

int outlier_detection(lua_State *L)
{
   THCState *state = getCutorchState(L);
   THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
   THCudaTensor *d1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
   THCudaTensor *outlier = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
   int disp_max = luaL_checkinteger(L, 4);
   THCudaTensor *conf1 = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
   THCudaTensor *conf2 = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
   float t1 = luaL_checknumber(L, 7);
   float t2 = luaL_checknumber(L, 8);

   outlier_detection<<<(THCudaTensor_nElement(state, d0) - 1) / TB + 1, TB>>>(
      THCudaTensor_data(state, d0),
      THCudaTensor_data(state, d1),
      THCudaTensor_data(state, outlier),
      THCudaTensor_nElement(state, d0),
      THCudaTensor_size(state, d0, 3),
      THCudaTensor_data(state, conf1),
      THCudaTensor_data(state, conf2),
      disp_max, t1, t2);
   checkCudaError(L);
   return 0;
}

__global__ void L2dist_(float *input_L, float *input_R, float *output_L, float *output_R, int size1_input, int size1, int size3, int size23)
{
   int id = blockIdx.x * blockDim.x + threadIdx.x;
   if (id < size23) {
      int dim3 = id % size3;
      assert(size1_input <= 512);
      float L_cache[512];
      for (int i = 0; i < size1_input; i++) {
         L_cache[i] = input_L[i * size23 + id];
      }

      for (int d = 0; d < size1; d++) {
         if (dim3 - d >= 0) {
            float sum = 0;
            float diff = 0;
            for (int i = 0; i < size1_input; i++) {
               diff = L_cache[i] - input_R[i * size23 + id - d];
               sum += diff*diff;
            }
            sum = sqrt(sum);
            output_L[d * size23 + id] = sum;
            output_R[d * size23 + id - d] = sum;
         }
      }
   }
}

int L2dist(lua_State *L)
{
   THCState *state = getCutorchState(L);
   THCudaTensor *input_L = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
   THCudaTensor *input_R = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
   THCudaTensor *output_L = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
   THCudaTensor *output_R = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
   int size23 = THCudaTensor_size(state, output_L, 2) * THCudaTensor_size(state, output_L, 3);
   L2dist_<<<(size23 - 1) / TB + 1, TB>>>(
      THCudaTensor_data(state, input_L),
      THCudaTensor_data(state, input_R),
      THCudaTensor_data(state, output_L),
      THCudaTensor_data(state, output_R),
      THCudaTensor_size(state, input_L, 1),
      THCudaTensor_size(state, output_L, 1),
      THCudaTensor_size(state, output_L, 3),
      size23);
   checkCudaError(L);
   return 0;
}

static const struct luaL_Reg funcs[] = {
   {"outlier_detection", outlier_detection},
   {"L2dist", L2dist},
   {NULL, NULL}
};

extern "C" int luaopen_libcuresmatch(lua_State *L) {
	srand(42);
	luaL_openlib(L, "curesmatch", funcs, 0);
   return 1;
}
