#include "hip/hip_runtime.h"
extern "C" {
   #include "lua.h"
   #include "lualib.h"
   #include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include "hip/hip_runtime.h"
#include "npp.h"

#define TB 128

#define NPP_CALL(x) {const NppStatus a = (x); if (a != NPP_SUCCESS) {printf("\nNPP Error: (err_num=%d) \n", a);} }

THCState* getCutorchState(lua_State* L)
{
	lua_getglobal(L, "cutorch");
	lua_getfield(L, -1, "getState");
	lua_call(L, 0, 1);
	THCState *state = (THCState*) lua_touserdata(L, -1);
	lua_pop(L, 2);
	return state;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

THCudaTensor *new_tensor_like(THCState *state, THCudaTensor *x)
{
	THCudaTensor *y = THCudaTensor_new(state);
	THCudaTensor_resizeAs(state, y, x);
	return y;
}

__global__ void depth_filter(float *img, float*out, int size, int height, int width, int threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0)
	{
		printf("Size is %d\n", size);
		printf("Height is %d\n", height);
		printf("Width is %d\n", width);
		printf("Threshold is %d\n", threshold);
		printf("Img first value is %f\n", img[0]);
	}
	if(id < size)
	{
		//int col = id % width;
		//int row = id / width;
		if(img[id] < threshold)
		{
			out[id] = 0;
		}
		else
		{
			out[id] = img[id];
		}
	}
}

int depth_filter(lua_State *L)
{
	printf("Entering depth_filter\n");
	THCState *state = getCutorchState(L);
	printf("Got the state\n");
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	printf("Got the img\n");
	int threshold = luaL_checknumber(L, 2);
	printf("Got the threshold\n");
	THCudaTensor *out = new_tensor_like(state, img);
	printf("Made the out tensor\n");
	
	depth_filter<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
			THCudaTensor_data(state, img),
			THCudaTensor_data(state, out),
			THCudaTensor_nElement(state, out),
			THCudaTensor_size(state, out, 2),
			THCudaTensor_size(state, out, 3),
			threshold);
	printf("Executed depth_filter\n");
	checkCudaError(L);
	printf("Checked cuda error\n");
	luaT_pushudata(L, out, "torch.CudaTensor");
	printf("Pushed data\n");
	return 1;
}

int erode(lua_State *L)
{
	printf("Entered to ERODE method\n");
	
	THCState *state = getCutorchState(L);
	
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *kernel = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = new_tensor_like(state, img);
	
	printf("Parameters read correctly\n");
	
	long half_kernel_width = (kernel->size[0] - 1) / 2;
	long int y = img->size[2];
	long int x = img->size[3];
	long int c = img->size[1];
	
	printf("Dimensions retrieved correctly: y:%d, x:%d, c:%d, half kernel:%d\n", y, x, c, half_kernel_width);
			
	NppiSize oSizeRoi;
	oSizeRoi.width = x - (half_kernel_width * 2);
	oSizeRoi.height = y - (half_kernel_width * 2);
	
	//oSizeRoi.width = 3;
	//oSizeRoi.height = 1;
	
	printf("ROI size generated: width:%d, height:%d\n", oSizeRoi.width, oSizeRoi.height);
	
	NppiSize oMaskSize;
	oMaskSize.width = kernel->size[0];
	oMaskSize.height = kernel->size[1];
	
	printf("Mask size generated: width:%d, height:%d\n", oMaskSize.width, oMaskSize.height);
	
	NppiPoint oAnchor;
	oAnchor.x = half_kernel_width;
	oAnchor.y = half_kernel_width;
	
	printf("Anchor point generated: x:%d, y:%d\n", oAnchor.x, oAnchor.y);
	
	printf("Test accesing to an element of the img. Element 0: %f\n", &(img->storage->data)[1242]);
	printf("Test size of the img data. Element 0: %f\n", sizeof(&(img->storage->data)));
	printf("Test size to an element of the img. Element 0: %d\n", sizeof(typeof(&(img->storage->data)[1242])));
	
	Npp32f *pSrc = img->storage->data + img->stride[2] * half_kernel_width + img->stride[3] * half_kernel_width;
	
	Npp32f *pDst = out->storage->data + out->stride[0] * half_kernel_width + out->stride[1] * half_kernel_width;
	
	
	Npp8u pMask[9];
		
	/*float *kernel_data = THCudaTensor_data(state, kernel);
	
	for(int i = 0; i < oMaskSize.width; i++)
	{
		printf("Reading kernel data %f\n", kernel_data[i]);
		pMask[i] = static_cast<unsigned int>(kernel_data[i]);
	}*/
	
	pMask[0] = 0;
	pMask[1] = 1;
	pMask[2] = 0;
	pMask[3] = 1;
	pMask[4] = 1;
	pMask[5] = 1;
	pMask[6] = 0;
	pMask[7] = 1;
	pMask[8] = 0;
	
	
	
	printf("Checking step sizes: %d, %d\n", sizeof(Npp32f) * img->stride[2], sizeof(Npp32f) * out->stride[3]);
	
	NPP_CALL(nppiErode_32f_C1R(pSrc, sizeof(Npp32f) * (img->stride[2] - 2), out->storage->data, sizeof(Npp32f) * img->stride[2], oSizeRoi, pMask, oMaskSize, oAnchor));
	
	printf("nppiErode executed...\n");
	
	
	checkCudaError(L);
	
	luaT_pushudata(L, out, "torch.CudaTensor");
	
	return 1;
}

static const struct luaL_Reg funcs[] = {
	{"depth_filter", depth_filter},
	{"erode", erode},
	{NULL, NULL}
};

extern "C" int luaopen_libgdcutils(lua_State *L) {
	srand(42);
	luaL_openlib(L, "gdcutils", funcs, 0);
	return 1;
}